#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN

#include "cudnn_full_conv2d.h"
#include "doctest.h"
#include <hipDNN.h>

TEST_CASE( "Check cudnn full convolution" ) {

  const size_t dN = 1, dC = 16, dH = 32, dW = 32, dF = 16, dKH = 3, dKW = 3;
  float *      U, V;
  void*        K;

  hipMalloc( &U, ( dN * dC * dH * dW ) * sizeof( float ) );
  hipMalloc( &V, ( dN * dC * dH * dW ) * sizeof( float ) );
  hipMalloc( &K, ( dC * dF * dKH * dKW ) * sizeof( float ) );

  CHECK(cudnn_imp::conv2d(U, K, V, dN, dC, dH, dW, dF, dKH, dKW)
      == )

  hipFree( U );
  hipFree( V );
  hipFree( K );
}
