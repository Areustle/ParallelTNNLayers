#include "Tensor.h"

using namespace std;

Tensor::Tensor(size_t N, size_t C, size_t H, size_t W)
    : N(N)
    , C(C)
    , H(H)
    , W(W)
    , len(N * C * H * W) {
  hipMallocManaged(&m_data, len * sizeof(float));
  hipMemset(&m_data, 0, len);
  hipDeviceSynchronize();
}

Tensor::Tensor(Tensor const& other)
    : N(other.N)
    , C(other.C)
    , H(other.H)
    , W(other.W)
    , len(other.len) {
  hipMallocManaged(&m_data, len * sizeof(float));
  hipMemcpy(
      m_data, other.m_data, len * sizeof(float), hipMemcpyDeviceToDevice);
  hipDeviceSynchronize();
}

Tensor& Tensor::operator=(Tensor const& other) {
  if (this == &other)
    return *this;
  if (len != other.len) {
    delete[] m_data;
    len = other.len;
    hipMallocManaged(&m_data, len * sizeof(float));
  }
  hipMemcpy(
      m_data, other.m_data, len * sizeof(float), hipMemcpyDeviceToDevice);
  hipDeviceSynchronize();
  return *this;
}

Tensor::~Tensor() {
  hipDeviceSynchronize();
  hipFree(m_data);
}
