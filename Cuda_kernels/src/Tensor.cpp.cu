#include "Tensor.h"

#include "CudaAllocator.h"


Tensor::Tensor(size_t N, size_t C, size_t H, size_t W)
    : N(N)
    , C(C)
    , H(H)
    , W(W)
    , size(N * C * H * W) {
  hipMallocManaged(&m_data, size * sizeof(float));
  hipMemset(&m_data, 0, size);
  hipDeviceSynchronize();
}

Tensor::Tensor(Tensor const& other)
    : N(other.N)
    , C(other.C)
    , H(other.H)
    , W(other.W)
    , size(other.size) {
  hipMallocManaged(&m_data, size * sizeof(float));
  hipMemcpy(m_data, other.m_data, size * sizeof(float), hipMemcpyDeviceToDevice);
  hipDeviceSynchronize();
}

Tensor& Tensor::operator=(Tensor const& other) {
  if (this == &other)
    return *this;
  if (size != other.size) {
    delete[] m_data;
    size = other.size;
    hipMallocManaged(&m_data, size * sizeof(float));
  }
  hipMemcpy(m_data, other.m_data, size * sizeof(float), hipMemcpyDeviceToDevice);
  hipDeviceSynchronize();
  return *this;
}

Tensor::~Tensor() {
  hipDeviceSynchronize();
  hipFree(m_data);
}
