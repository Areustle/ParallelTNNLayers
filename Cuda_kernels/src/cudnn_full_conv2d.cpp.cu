#include "cudnn_full_conv2d.h"
#include <hipDNN.h>

Tensor nn_conv2d(Tensor const U, Tensor const K) {

  void*                        d_workspace;
  hipdnnHandle_t                cudnn;
  hipdnnTensorDescriptor_t      input_descriptor;
  hipdnnTensorDescriptor_t      output_descriptor;
  hipdnnFilterDescriptor_t      kernel_descriptor;
  hipdnnConvolutionDescriptor_t convolution_descriptor;
  hipdnnConvolutionFwdAlgo_t    convolution_algorithm;
  size_t                       workspace_bytes = 0;

  hipdnnCreate(&cudnn);
  hipdnnCreateTensorDescriptor(&input_descriptor);
  hipdnnSetTensor4dDescriptor(
      input_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, U.N, U.C, U.H, U.W);
  hipdnnCreateTensorDescriptor(&output_descriptor);
  hipdnnSetTensor4dDescriptor(
      output_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, U.N, U.C, U.H, U.W);
  hipdnnCreateFilterDescriptor(&kernel_descriptor);
  hipdnnSetFilter4dDescriptor(
      kernel_descriptor, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, K.N, K.C, K.H, K.W);
  hipdnnCreateConvolutionDescriptor(&convolution_descriptor);
  hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                  1,
                                  1,
                                  1,
                                  1,
                                  1,
                                  1,
                                  HIPDNN_CROSS_CORRELATION,
                                  HIPDNN_DATA_FLOAT);
  hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                      input_descriptor,
                                      kernel_descriptor,
                                      convolution_descriptor,
                                      output_descriptor,
                                      HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                      /*memoryLimitInBytes=*/0,
                                      &convolution_algorithm);
  hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                          input_descriptor,
                                          kernel_descriptor,
                                          convolution_descriptor,
                                          output_descriptor,
                                          convolution_algorithm,
                                          &workspace_bytes);

  hipMallocManaged(&d_workspace, workspace_bytes);

  Tensor V(U);

  const float alpha = 1, beta = 0;
  hipdnnConvolutionForward(cudnn,
                          &alpha,
                          input_descriptor,
                          U.m_data,
                          kernel_descriptor,
                          K.m_data,
                          convolution_descriptor,
                          convolution_algorithm,
                          d_workspace,
                          workspace_bytes,
                          &beta,
                          output_descriptor,
                          V.m_data);

  hipDeviceSynchronize();
  hipFree(d_workspace);
  hipdnnDestroyTensorDescriptor(input_descriptor);
  hipdnnDestroyTensorDescriptor(output_descriptor);
  hipdnnDestroyFilterDescriptor(kernel_descriptor);
  hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
  hipdnnDestroy(cudnn);

  return V;
}


#define DOCTEST_CONFIG_IMPLEMENTATION_IN_DLL
#include "doctest.h"

TEST_CASE("cudnn_full_conv2d test") {
  Tensor U(1, 1, 32, 32);
  Tensor K(1, 1, 3, 3);

  auto V = nn_conv2d(U,K);

  for (int i = 0; i < V.size(); ++i) {
    CHECK(V[i] == 0);
  }
}
