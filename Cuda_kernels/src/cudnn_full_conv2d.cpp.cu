#include "Cuda_kernels/cudnn_full_conv2d.h"
#include <hipDNN.h>
#include <iostream>


CudnnConv2d::CudnnConv2d(const size_t N,
                         const size_t C,
                         const size_t H,
                         const size_t W,
                         const size_t F,
                         const size_t Y,
                         const size_t X)
    : N(N)
    , C(C)
    , H(H)
    , W(W)
    , F(F)
    , Y(Y)
    , X(X) {

  hipdnnCreate(&cudnn);
  hipdnnCreateTensorDescriptor(&input_descriptor);
  hipdnnSetTensor4dDescriptor(
      input_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, H, W);
  hipdnnCreateTensorDescriptor(&output_descriptor);
  hipdnnSetTensor4dDescriptor(
      output_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, H, W);
  hipdnnCreateFilterDescriptor(&kernel_descriptor);
  hipdnnSetFilter4dDescriptor(
      kernel_descriptor, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, C, F, Y, X);
  hipdnnCreateConvolutionDescriptor(&convolution_descriptor);
  hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                  1,
                                  1,
                                  1,
                                  1,
                                  1,
                                  1,
                                  HIPDNN_CROSS_CORRELATION,
                                  HIPDNN_DATA_FLOAT);
  hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                      input_descriptor,
                                      kernel_descriptor,
                                      convolution_descriptor,
                                      output_descriptor,
                                      HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                      /*memoryLimitInBytes=*/0,
                                      &convolution_algorithm);
  hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                          input_descriptor,
                                          kernel_descriptor,
                                          convolution_descriptor,
                                          output_descriptor,
                                          convolution_algorithm,
                                          &workspace_bytes);

  hipMalloc(&d_workspace, workspace_bytes);
}

CudnnConv2d::~CudnnConv2d() {
  hipFree(d_workspace);
  hipdnnDestroyTensorDescriptor(input_descriptor);
  hipdnnDestroyTensorDescriptor(output_descriptor);
  hipdnnDestroyFilterDescriptor(kernel_descriptor);
  hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
  hipdnnDestroy(cudnn);
}


void CudnnConv2d::opearator()(float* Input, float* Kernel, float* Output) {
  const float alpha = 1, beta = 0;
  hipdnnConvolutionForward(cudnn,
                          &alpha,
                          input_descriptor,
                          Input,
                          kernel_descriptor,
                          Kernel,
                          convolution_descriptor,
                          convolution_algorithm,
                          d_workspace,
                          workspace_bytes,
                          &beta,
                          output_descriptor,
                          Output);
}

