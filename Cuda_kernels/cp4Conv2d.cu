#include "hip/hip_runtime.h"
#include "cp4Conv2d.cuh"
#include <iostream>
#include <stdlib.h>

using namespace std;

__constant__ float const_filter[4096];

/*******************************************************************************
 * 2 Dimensional Convolution Operation using an order-4 CP decomposition.
 * Also known as a Candecomp/Parafac Decomposition, a Canonical Polyadic
 * Decomposition, and a Tensor Rank Decomposition.
 *******************************************************************************/
__global__ void conv2d_cp4_kernel(float* __restrict__ Out,
                                  const float* __restrict__ Input,
                                  const unsigned N,
                                  const unsigned C,
                                  const unsigned H,
                                  const unsigned W,
                                  const unsigned pad,
                                  const unsigned offset_fK,
                                  const unsigned offset_fC,
                                  const unsigned offset_fH,
                                  const unsigned offset_fW,
                                  const unsigned Rank,
                                  const unsigned fK,
                                  const unsigned fC,
                                  const unsigned fH,
                                  const unsigned fW,
                                  const unsigned WgrdDim,
                                  const unsigned Bw,
                                  const unsigned Bh,
                                  const unsigned Bc,
                                  const unsigned iEnd,
                                  const unsigned jEnd,
                                  const unsigned sW,
                                  const unsigned sH) {

  extern __shared__ float shared_mem[];

  const unsigned w         = threadIdx.x % Bw;
  const unsigned h         = threadIdx.x / Bw;
  const unsigned wBlockOff = (blockIdx.x % WgrdDim) * Bw;
  const unsigned hBlockOff = (blockIdx.x / WgrdDim) * Bh;

  // Grid Stride loop to handle overlarge batch (n) and filter (k) sizes
  for (unsigned n = blockIdx.z / fK; n < N; n += blockDim.z * gridDim.z) {
    for (unsigned k = blockIdx.z % fK; k < fK; k += blockDim.z * gridDim.z) {

      float partial_channel_sum = 0.0f;

      for (unsigned c = threadIdx.y; c < C; c += blockDim.y) {
        // Shift the Global pointers to our Region Of interest
        const float* iPtr = Input + n * C * H * W + c * H * W;
        float*       sPtr = shared_mem + threadIdx.y * sH * sW;

        // clang-format off
        // Cooperatively load all input segment into our shared memory and pad it.
        for (unsigned j = h; j < jEnd; j += Bh)  // For every participating h pixel
        for (unsigned i = w; i < iEnd; i += Bw)  // For every participating w pixel
          sPtr[j*sW + i] = (j+hBlockOff >= pad
                && j+hBlockOff < H+pad
                && i+wBlockOff >= pad
                && i+wBlockOff < W+pad)
            ?(iPtr[(j+hBlockOff-pad)*W         // Height
                  + (i+wBlockOff-pad)])  // Width
            :(0.0f); // Pad with Zeros if outside the bounds

        __syncthreads();
        // clang-format on

        // Handle block / input size mismatch. This occurs here and not earlier
        // So that these threads can still participate in the cooperative shared
        // Memory load.
        if (hBlockOff + h >= H) continue;
        if (wBlockOff + w >= W) continue;

        float pixel_sum = 0.0f;

        // Perform Convolution from shared memory.
        // Accumulate sum of products in 'pixel_sum' variable.
        for (unsigned rr = 0; rr < Rank; ++rr) {

          // Store intermediate results for each rank.
          float rank_sum = 0.0f;

          // sum of products for filter height and width.
          for (unsigned fh = 0; fh < fH; ++fh)
            for (unsigned fw = 0; fw < fW; ++fw)
              rank_sum += sPtr[(h + fh) * sW + (w + fw)]
                          * const_filter[offset_fH + fh * Rank + rr]
                          * const_filter[offset_fW + fw * Rank + rr];

          // Avoid redundant work in nested loop.
          rank_sum *= const_filter[offset_fK + k * Rank + rr]
                      * const_filter[offset_fC + c * Rank + rr];

          // accumulate pixel value for this channel.
          pixel_sum += rank_sum;
        }

        __syncthreads();
        sPtr[h * sW + w] = pixel_sum;
        __syncthreads();

        for (unsigned cc = blockDim.y / 2; cc > 0; cc >>= 1) {
          if (threadIdx.y < cc)
            shared_mem[threadIdx.y * sH * sW + h * sW + w]
                += shared_mem[(threadIdx.y + cc) * sH * sW + h * sW + w];
          __syncthreads();
        }

        partial_channel_sum += shared_mem[h * sW + w];
      }

      // populate output array.
      if (threadIdx.y == 0)
        Out[n * fK * H * W + k * H * W + (h + hBlockOff) * W + w + wBlockOff]
            = partial_channel_sum;
    }
  }
}


void cuda_conv2d_cp4_gpu(const float*   In,
                         const unsigned N,
                         const unsigned C,
                         const unsigned H,
                         const unsigned W,
                         const unsigned pad,
                         const float*   FilterK,
                         const float*   FilterC,
                         const float*   FilterH,
                         const float*   FilterW,
                         const unsigned fRank,
                         const unsigned fK,
                         const unsigned fC,
                         const unsigned fH,
                         const unsigned fW,
                         float*         Out) {

  // This implementation uses the GPU's constant memory as a fast cache to
  // hold the relatively small and unchanging filter weights. These must all
  // be accessed uniformly by the threads in a block for parallel execution.
  // Populate GPU constant memory with the 4 filters at an appropriate offset.
  const unsigned offset_fK = 0;
  const unsigned offset_fC = offset_fK + (fK * fRank);
  const unsigned offset_fH = offset_fC + (fC * fRank);
  const unsigned offset_fW = offset_fH + (fH * fRank);
  hipMemcpyToSymbol(HIP_SYMBOL(const_filter),
                     FilterK,
                     sizeof(float) * (fK * fRank),
                     sizeof(float) * offset_fK);
  hipMemcpyToSymbol(HIP_SYMBOL(const_filter),
                     FilterC,
                     sizeof(float) * (fC * fRank),
                     sizeof(float) * offset_fC);
  hipMemcpyToSymbol(HIP_SYMBOL(const_filter),
                     FilterH,
                     sizeof(float) * (fH * fRank),
                     sizeof(float) * offset_fH);
  hipMemcpyToSymbol(HIP_SYMBOL(const_filter),
                     FilterW,
                     sizeof(float) * (fW * fRank),
                     sizeof(float) * offset_fW);

  const unsigned Bh   = 8;
  const unsigned Bw   = 32;
  const unsigned Bc   = 2;
  const size_t   smsz = Bc            //
                      * (fW - 1 + Bw) //
                      * (fH - 1 + Bh) //
                      * sizeof(float);

  const unsigned WgrdDim = (W / Bw) + ((W % Bw) != 0);
  const unsigned HgrdDim = (H / Bh) + ((H % Bh) != 0);
  const dim3     Gshp(WgrdDim * HgrdDim, 1, fK * N);
  const dim3     Bshp(Bw * Bh, Bc, 1);
  const unsigned iEnd = fW - 1 + Bw;
  const unsigned jEnd = fH - 1 + Bh;
  const unsigned sW   = fW - 1 + Bw;
  const unsigned sH   = fH - 1 + Bh;

  conv2d_cp4_kernel<<<Gshp, Bshp, smsz>>>(Out,
                                          In,
                                          N,
                                          C,
                                          H,
                                          W,
                                          pad,
                                          offset_fK,
                                          offset_fC,
                                          offset_fH,
                                          offset_fW,
                                          fRank,
                                          fK,
                                          fC,
                                          fH,
                                          fW,
                                          WgrdDim,
                                          Bw,
                                          Bh,
                                          Bc,
                                          iEnd,
                                          jEnd,
                                          sW,
                                          sH);
  hipDeviceSynchronize();
}


Tensor conv2d_cp4_gpu(Tensor const Input,
                      Tensor const FilterK,
                      Tensor const FilterC,
                      Tensor const FilterH,
                      Tensor const FilterW,
                      int          pad) {

  const int N     = Input.shape[0];
  const int C     = Input.shape[1];
  const int H     = Input.shape[2];
  const int W     = Input.shape[3];
  const int fRank = FilterK.shape[1];
  const int fK    = FilterK.shape[0];
  const int fC    = FilterC.shape[0];
  const int fH    = FilterH.shape[0];
  const int fW    = FilterW.shape[0];

  Tensor Out{ N, fK, H, W };
  cuda_conv2d_cp4_gpu(Input.m_data,
                      N,
                      C,
                      H,
                      W,
                      pad,
                      FilterK.m_data,
                      FilterC.m_data,
                      FilterH.m_data,
                      FilterW.m_data,
                      fRank,
                      fK,
                      fC,
                      fH,
                      fW,
                      Out.m_data);

  return Out;
}

Tensor conv2d_cp4_cpu(Tensor const Input,
                      Tensor const FilterK,
                      Tensor const FilterC,
                      Tensor const FilterR,
                      Tensor const FilterS,
                      int          pad) {

  const int N    = Input.shape[0];
  const int C    = Input.shape[1];
  const int iH   = Input.shape[2];
  const int oH   = iH - 2 * pad;
  const int iW   = Input.shape[3];
  const int oW   = iW - 2 * pad;
  const int Rank = FilterK.shape[1];
  const int fK   = FilterK.shape[0];
  const int fC   = FilterC.shape[0];
  const int fH   = FilterR.shape[0];
  const int fW   = FilterS.shape[0];

  Tensor Out{ N, C, oH, oW };

  // clang-format off
  for (int n = 0; n < N; ++n)
  for (int k = 0; k < fK; ++k)
  for (int h = 0; h < oH; ++h)
  for (int w = 0; w < oW; ++w){
    float sum = 0.0f;
    for (int c = 0; c < C; ++c)
    for (int rr = 0; rr < Rank; ++rr)
    for (int fh = 0; fh < fH; ++fh)
    for (int fw = 0; fw < fW; ++fw){
      sum += Input.m_data[n*C*iH*iW + c*iH*iW + (h+fh)*iW + w+fw]
      *  FilterK.m_data[k*Rank + rr]
      *  FilterC.m_data[c*Rank + rr]
      *  FilterR.m_data[fh*Rank + rr]
      *  FilterS.m_data[fw*Rank + rr];
    }
    Out.m_data[n*C*oH*oW + k*oH*oW + h*oW + w] = sum;
  }
  // clang-format on
  return Out;
}


int main(int argc, char** argv) {

  unsigned N     = 1;
  unsigned C     = 16;
  unsigned H     = 32;
  unsigned W     = 32;
  unsigned pad   = 1;
  unsigned fK    = 16;
  unsigned fH    = 3;
  unsigned fW    = 3;
  unsigned fRank = 1;

  if (argc != 10)
    cerr << "Using Default shape" << endl;
  else {
    N     = atoi(argv[1]);
    C     = atoi(argv[2]);
    H     = atoi(argv[3]);
    W     = atoi(argv[4]);
    pad   = atoi(argv[5]);
    fK    = atoi(argv[6]);
    fH    = atoi(argv[7]);
    fW    = atoi(argv[8]);
    fRank = atoi(argv[9]);
  }


  float* In;
  float* Out;
  float* FilterK;
  float* FilterC;
  float* FilterW;
  float* FilterH;

  hipMalloc(&In, N * C * H * W * sizeof(float));
  hipMalloc(&FilterK, fK * fRank * sizeof(float));
  hipMalloc(&FilterC, C * fRank * sizeof(float));
  hipMalloc(&FilterH, fH * fRank * sizeof(float));
  hipMalloc(&FilterW, fW * fRank * sizeof(float));
  hipMalloc(&Out, N * fK * H * W * sizeof(float));

  cuda_conv2d_cp4_gpu(In,
                      N,
                      C,
                      H,
                      W,
                      pad,
                      FilterK,
                      FilterC,
                      FilterH,
                      FilterW,
                      fRank,
                      fK,
                      C,
                      fH,
                      fW,
                      Out);


  hipFree(In);
  hipFree(FilterK);
  hipFree(FilterC);
  hipFree(FilterH);
  hipFree(FilterW);
  hipFree(Out);
}
