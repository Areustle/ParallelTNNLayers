#include "cudnn_full_conv2d.h"
#include <hipDNN.h>
#include <iostream>

#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

Tensor nn_conv2d(Tensor const U, Tensor const K) {

  hipdnnHandle_t cudnn;
  hipdnnCreate(&cudnn);

  hipdnnTensorDescriptor_t input_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        U.shape[0],
                                        U.shape[1],
                                        U.shape[2],
                                        U.shape[3]));

  hipdnnFilterDescriptor_t kernel_descriptor;
  checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
  checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                        HIPDNN_DATA_FLOAT,
                                        HIPDNN_TENSOR_NCHW,
                                        K.shape[0],
                                        K.shape[1],
                                        K.shape[2],
                                        K.shape[3]));

  hipdnnConvolutionDescriptor_t convolution_descriptor;
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                             /*pad_height=*/1,
                                             /*pad_width=*/1,
                                             /*vertical_stride=*/1,
                                             /*horizontal_stride=*/1,
                                             /*dilation_height=*/1,
                                             /*dilation_width=*/1,
                                             /*mode=*/HIPDNN_CROSS_CORRELATION,
                                             /*computeType=*/HIPDNN_DATA_FLOAT));

  int batch_size{ 0 }, channels{ 0 }, height{ 0 }, width{ 0 };
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor,
                                                   input_descriptor,
                                                   kernel_descriptor,
                                                   &batch_size,
                                                   &channels,
                                                   &height,
                                                   &width));

  hipdnnTensorDescriptor_t output_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        batch_size,
                                        channels,
                                        height,
                                        width));


  hipdnnConvolutionFwdAlgo_t convolution_algorithm;
  checkCUDNN(
      hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                          input_descriptor,
                                          kernel_descriptor,
                                          convolution_descriptor,
                                          output_descriptor,
                                          HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                          /*memoryLimitInBytes=*/0,
                                          &convolution_algorithm));

  size_t workspace_bytes{ 0 };
  checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                     input_descriptor,
                                                     kernel_descriptor,
                                                     convolution_descriptor,
                                                     output_descriptor,
                                                     convolution_algorithm,
                                                     &workspace_bytes));

  void* d_workspace{ nullptr };
  hipMallocManaged(&d_workspace, workspace_bytes);

  /* size_t image_bytes = batch_size * channels * height * width *
   * sizeof(float); */

  float* d_input{ nullptr };
  hipMalloc(&d_input, U.size());
  hipMemcpy(d_input, U.m_data, U.size(), hipMemcpyHostToDevice);

  float* d_output{ nullptr };
  hipMalloc(&d_output, U.size());
  hipMemset(d_output, 0, U.size());

  Tensor V({ batch_size, channels, height, width });

  const float alpha = 1, beta = 0;
  hipdnnConvolutionForward(cudnn,
                          &alpha,
                          input_descriptor,
                          U.m_data,
                          kernel_descriptor,
                          K.m_data,
                          convolution_descriptor,
                          convolution_algorithm,
                          d_workspace,
                          workspace_bytes,
                          &beta,
                          output_descriptor,
                          V.m_data);

  hipDeviceSynchronize();
  hipFree(d_workspace);
  hipdnnDestroyTensorDescriptor(input_descriptor);
  hipdnnDestroyTensorDescriptor(output_descriptor);
  hipdnnDestroyFilterDescriptor(kernel_descriptor);
  hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
  hipdnnDestroy(cudnn);
  hipDeviceSynchronize();

  return V;
}

#define DOCTEST_CONFIG_IMPLEMENTATION_IN_DLL
#include "../external/doctest/doctest.h"
#include <random>

TEST_CASE("cudnn_full_conv2d test") {
  Tensor U{ 1, 1, 32, 32 };
  Tensor K{ 1, 1, 3, 3 };

  std::random_device               rd;
  std::mt19937                     gen(rd());
  std::uniform_real_distribution<> dis(0.1, 1.0);

  for (size_t i = 0; i < U.size(); ++i) U[i] = dis(gen);
  for (int i = 0; i < U.size(); ++i) CHECK(U[i] != 0);
  for (int i = 0; i < K.size(); ++i) CHECK(K[i] == 0);

  auto V = nn_conv2d(U, K);

  for (int i = 0; i < V.size(); ++i) CHECK(V[i] == 0);
}
