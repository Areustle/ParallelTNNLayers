#include "Tensor.h"

#include <numeric>

using namespace std;

Tensor::Tensor(std::initializer_list<int> l)
    : shape(l) {
  const int len =
      std::accumulate(shape.begin(), shape.end(), 1, std::multiplies<int>());
  hipMallocManaged(&m_data, len * sizeof(float));
  hipMemset(&m_data, 0, len);
  hipDeviceSynchronize();
}


Tensor::Tensor(Tensor const& other)
    : shape(other.shape) {
  const int len =
      std::accumulate(shape.begin(), shape.end(), 1, std::multiplies<int>());
  hipMallocManaged(&m_data, len * sizeof(float));
  hipMemcpy(
      m_data, other.m_data, len * sizeof(float), hipMemcpyDeviceToDevice);
  hipDeviceSynchronize();
}


Tensor& Tensor::operator=(const Tensor& other) {
  if (this == &other)
    return *this;
  if (this->size() != other.size()) {
    delete[] m_data;
    hipMallocManaged(&m_data, other.size() * sizeof(float));
  }
  hipMemcpy(m_data,
             other.m_data,
             other.size() * sizeof(float),
             hipMemcpyDeviceToDevice);
  hipDeviceSynchronize();
  return *this;
}


Tensor::~Tensor() {
  hipDeviceSynchronize();
  hipFree(m_data);
}

size_t Tensor::size() const {
  return std::accumulate(shape.begin(), shape.end(), 1, std::multiplies<int>());
}
