#include "Tensor.h"

#include "CudaAllocator.h"


Tensor::Tensor(size_t N, size_t C, size_t H, size_t W)
    : N(N)
    , C(C)
    , H(H)
    , W(W)
    , len(N * C * H * W) {
  hipMalloc(&data, len * sizeof(float));
  hipMemset(&data, 0, len);
  hipDeviceSynchronize();
}

Tensor::Tensor(Tensor const& other)
    : N(other.N)
    , C(other.C)
    , H(other.H)
    , W(other.W)
    , len(other.len) {
  hipMalloc(&data, len * sizeof(float));
  hipMemcpy(data, other.data, len, hipMemcpyDeviceToDevice);
  hipDeviceSynchronize();
}

Tensor& Tensor::operator=(Tensor const& other) {
  if (this == &other)
    return *this;
  if (len != other.len) {
    delete[] data;
    len = other.len;
    hipMalloc(&data, len * sizeof(float));
  }
  hipMemcpy(data, other.data, len, hipMemcpyDeviceToDevice);
  hipDeviceSynchronize();
  return *this;
}

Tensor::~Tensor() {
  hipDeviceSynchronize();
  hipFree(data);
}
