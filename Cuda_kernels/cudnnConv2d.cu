#include "cudnnConv2d.cuh"
#include <hipDNN.h>
#include <iostream>

#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

Tensor nn_conv2d(Tensor const U, Tensor const K) {

  hipdnnHandle_t cudnn;
  hipdnnCreate(&cudnn);

  hipdnnTensorDescriptor_t input_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        U.shape[0],
                                        U.shape[1],
                                        U.shape[2],
                                        U.shape[3]));

  hipdnnFilterDescriptor_t kernel_descriptor;
  checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
  checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                        HIPDNN_DATA_FLOAT,
                                        HIPDNN_TENSOR_NCHW,
                                        K.shape[0],
                                        K.shape[1],
                                        K.shape[2],
                                        K.shape[3]));

  hipdnnConvolutionDescriptor_t convolution_descriptor;
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                             /*pad_height=*/1,
                                             /*pad_width=*/1,
                                             /*vertical_stride=*/1,
                                             /*horizontal_stride=*/1,
                                             /*dilation_height=*/1,
                                             /*dilation_width=*/1,
                                             /*mode=*/HIPDNN_CROSS_CORRELATION,
                                             /*computeType=*/HIPDNN_DATA_FLOAT));

  int batch_size{ 0 }, channels{ 0 }, height{ 0 }, width{ 0 };
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor,
                                                   input_descriptor,
                                                   kernel_descriptor,
                                                   &batch_size,
                                                   &channels,
                                                   &height,
                                                   &width));

  hipdnnTensorDescriptor_t output_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        batch_size,
                                        channels,
                                        height,
                                        width));


  hipdnnConvolutionFwdAlgo_t convolution_algorithm;
  checkCUDNN(
      hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                          input_descriptor,
                                          kernel_descriptor,
                                          convolution_descriptor,
                                          output_descriptor,
                                          HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                          /*memoryLimitInBytes=*/0,
                                          &convolution_algorithm));

  size_t workspace_bytes{ 0 };
  checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                     input_descriptor,
                                                     kernel_descriptor,
                                                     convolution_descriptor,
                                                     output_descriptor,
                                                     convolution_algorithm,
                                                     &workspace_bytes));

  void* d_workspace{ nullptr };
  hipMallocManaged(&d_workspace, workspace_bytes);

  float* d_input{ nullptr };
  hipMalloc(&d_input, U.size());
  hipMemcpy(d_input, U.m_data, U.size(), hipMemcpyHostToDevice);

  size_t out_bytes = batch_size * channels * height * width * sizeof(float);
  float* d_output{ nullptr };
  hipMalloc(&d_output, out_bytes);
  hipMemset(d_output, 0, out_bytes);

  Tensor V({ batch_size, channels, height, width });

  const float alpha = 1, beta = 0;
  hipdnnConvolutionForward(cudnn,
                          &alpha,
                          input_descriptor,
                          U.m_data,
                          kernel_descriptor,
                          K.m_data,
                          convolution_descriptor,
                          convolution_algorithm,
                          d_workspace,
                          workspace_bytes,
                          &beta,
                          output_descriptor,
                          V.m_data);

  hipFree(d_workspace);
  hipdnnDestroyTensorDescriptor(input_descriptor);
  hipdnnDestroyTensorDescriptor(output_descriptor);
  hipdnnDestroyFilterDescriptor(kernel_descriptor);
  hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
  hipdnnDestroy(cudnn);

  return V;
}
