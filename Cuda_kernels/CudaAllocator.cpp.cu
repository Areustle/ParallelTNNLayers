#include "CudaAllocator.h"

float* CudaAllocator::allocate(std::size_t n) {
  float* p;
  hipMallocManaged(&p, n * sizeof(float));
  hipDeviceSynchronize();
  return p;
}

void CudaAllocator::deallocate(float* p, std::size_t n) {
  hipDeviceSynchronize();
  hipFree(p);
}


