#include "cudnn_full_conv2d.h"
#include <hipDNN.h>
#include <iostream>

void cudnn_imp::conv2d( float*       U,
                        void*        K,
                        float*       V,
                        const size_t dN,
                        const size_t dC,
                        const size_t dH,
                        const size_t dW,
                        const size_t dF,
                        const size_t dKH,
                        const size_t dKW ) {

  void* d_workspace;

  hipdnnHandle_t                cudnn;
  hipdnnTensorDescriptor_t      input_descriptor;
  hipdnnTensorDescriptor_t      output_descriptor;
  hipdnnFilterDescriptor_t      kernel_descriptor;
  hipdnnConvolutionDescriptor_t convolution_descriptor;
  hipdnnConvolutionFwdAlgo_t    convolution_algorithm;
  size_t                       workspace_bytes = 0;

  hipdnnCreate( &cudnn );
  hipdnnCreateTensorDescriptor( &input_descriptor );
  hipdnnSetTensor4dDescriptor(
      input_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, dN, dC, dH, dW );
  hipdnnCreateTensorDescriptor( &output_descriptor );
  hipdnnSetTensor4dDescriptor(
      output_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, dN, dC, dH, dW );
  hipdnnCreateFilterDescriptor( &kernel_descriptor );
  hipdnnSetFilter4dDescriptor( kernel_descriptor,
                              HIPDNN_DATA_FLOAT,
                              HIPDNN_TENSOR_NCHW,
                              dC,
                              dF,
                              dKH,
                              dKW );
  hipdnnCreateConvolutionDescriptor( &convolution_descriptor );
  hipdnnSetConvolution2dDescriptor( convolution_descriptor,
                                   1,
                                   1,
                                   1,
                                   1,
                                   1,
                                   1,
                                   HIPDNN_CROSS_CORRELATION,
                                   HIPDNN_DATA_FLOAT );
  hipdnnGetConvolutionForwardAlgorithm( cudnn,
                                       input_descriptor,
                                       kernel_descriptor,
                                       convolution_descriptor,
                                       output_descriptor,
                                       HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                       /*memoryLimitInBytes=*/0,
                                       &convolution_algorithm );
  hipdnnGetConvolutionForwardWorkspaceSize( cudnn,
                                           input_descriptor,
                                           kernel_descriptor,
                                           convolution_descriptor,
                                           output_descriptor,
                                           convolution_algorithm,
                                           &workspace_bytes );
  /* cerr << "Workspace size: " << workspace_bytes << endl; */


  /* hipMalloc( &U, ( dN * dC * dH * dW ) * sizeof( float ) ); */
  /* hipMalloc( &V, ( dN * dC * dH * dW ) * sizeof( float ) ); */
  /* hipMalloc( &K, ( dC * dF * dKH * dKW ) * sizeof( float ) ); */

  hipMalloc( &d_workspace, workspace_bytes );

  const float alpha = 1, beta = 0;
  hipdnnConvolutionForward( cudnn,
                           &alpha,
                           input_descriptor,
                           U,
                           kernel_descriptor,
                           K,
                           convolution_descriptor,
                           convolution_algorithm,
                           d_workspace,
                           workspace_bytes,
                           &beta,
                           output_descriptor,
                           V );
  hipFree( d_workspace );

  /* hipFree( U ); */
  /* hipFree( V ); */
  /* hipFree( K ); */

  hipdnnDestroyTensorDescriptor( input_descriptor );
  hipdnnDestroyTensorDescriptor( output_descriptor );
  hipdnnDestroyFilterDescriptor( kernel_descriptor );
  hipdnnDestroyConvolutionDescriptor( convolution_descriptor );
  hipdnnDestroy( cudnn );
}
