#include "cudnn_full_conv2d.h"
#include <hipDNN.h>
#include <iostream>


CudnnConv2d::CudnnConv2d(const size_t N,
                         const size_t C,
                         const size_t H,
                         const size_t W,
                         const size_t F,
                         const size_t Y,
                         const size_t X)
    : N(N)
    , C(C)
    , H(H)
    , W(W)
    , F(F)
    , Y(Y)
    , X(X) {

  hipdnnCreate(&cudnn);
  hipdnnCreateTensorDescriptor(&input_descriptor);
  hipdnnSetTensor4dDescriptor(
      input_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, H, W);
  hipdnnCreateTensorDescriptor(&output_descriptor);
  hipdnnSetTensor4dDescriptor(
      output_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, H, W);
  hipdnnCreateFilterDescriptor(&kernel_descriptor);
  hipdnnSetFilter4dDescriptor(
      kernel_descriptor, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, C, F, Y, X);
  hipdnnCreateConvolutionDescriptor(&convolution_descriptor);
  hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                  1,
                                  1,
                                  1,
                                  1,
                                  1,
                                  1,
                                  HIPDNN_CROSS_CORRELATION,
                                  HIPDNN_DATA_FLOAT);
  hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                      input_descriptor,
                                      kernel_descriptor,
                                      convolution_descriptor,
                                      output_descriptor,
                                      HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                      /*memoryLimitInBytes=*/0,
                                      &convolution_algorithm);
  hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                          input_descriptor,
                                          kernel_descriptor,
                                          convolution_descriptor,
                                          output_descriptor,
                                          convolution_algorithm,
                                          &workspace_bytes);

  hipMalloc(&d_workspace, workspace_bytes);
}

CudnnConv2d::~CudnnConv2d() {
  hipFree(d_workspace);
  hipdnnDestroyTensorDescriptor(input_descriptor);
  hipdnnDestroyTensorDescriptor(output_descriptor);
  hipdnnDestroyFilterDescriptor(kernel_descriptor);
  hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
  hipdnnDestroy(cudnn);
}

void CudnnConv2d::conv2d(float* Input, float* Kernel, float* Output) {
  const float alpha = 1, beta = 0;
  hipdnnConvolutionForward(cudnn,
                          &alpha,
                          input_descriptor,
                          Input,
                          kernel_descriptor,
                          Kernel,
                          convolution_descriptor,
                          convolution_algorithm,
                          d_workspace,
                          workspace_bytes,
                          &beta,
                          output_descriptor,
                          Output);
}

/*void cudnn_imp::conv2d(float*       U, */
/*                       void*        K, */
/*                       float*       V, */
/*                       const size_t dN, */
/*                       const size_t dC, */
/*                       const size_t dH, */
/*                       const size_t dW, */
/*                       const size_t dF, */
/*                       const size_t dKH, */
/*                       const size_t dKW) { */

/*  void* d_workspace; */

/*  hipdnnHandle_t                cudnn; */
/*  hipdnnTensorDescriptor_t      input_descriptor; */
/*  hipdnnTensorDescriptor_t      output_descriptor; */
/*  hipdnnFilterDescriptor_t      kernel_descriptor; */
/*  hipdnnConvolutionDescriptor_t convolution_descriptor; */
/*  hipdnnConvolutionFwdAlgo_t    convolution_algorithm; */
/*  size_t                       workspace_bytes = 0; */

/*  hipdnnCreate(&cudnn); */
/*  hipdnnCreateTensorDescriptor(&input_descriptor); */
/*  hipdnnSetTensor4dDescriptor( */
/*      input_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, dN, dC, dH, dW);
 */
/*  hipdnnCreateTensorDescriptor(&output_descriptor); */
/*  hipdnnSetTensor4dDescriptor( */
/*      output_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, dN, dC, dH, dW);
 */
/*  hipdnnCreateFilterDescriptor(&kernel_descriptor); */
/*  hipdnnSetFilter4dDescriptor( */
/*      kernel_descriptor, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, dC, dF, dKH,
 * dKW); */
/*  hipdnnCreateConvolutionDescriptor(&convolution_descriptor); */
/*  hipdnnSetConvolution2dDescriptor(convolution_descriptor, */
/*                                  1, */
/*                                  1, */
/*                                  1, */
/*                                  1, */
/*                                  1, */
/*                                  1, */
/*                                  HIPDNN_CROSS_CORRELATION, */
/*                                  HIPDNN_DATA_FLOAT); */
/*  hipdnnGetConvolutionForwardAlgorithm(cudnn, */
/*                                      input_descriptor, */
/*                                      kernel_descriptor, */
/*                                      convolution_descriptor, */
/*                                      output_descriptor, */
/*                                      HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, */
/*                                      /1*memoryLimitInBytes=*/0,
    * /
        /*                                      &convolution_algorithm); */
        /*  hipdnnGetConvolutionForwardWorkspaceSize(cudnn, */
        /*                                          input_descriptor, */
        /*                                          kernel_descriptor, */
        /*                                          convolution_descriptor, */
        /*                                          output_descriptor, */
        /*                                          convolution_algorithm, */
        /*                                          &workspace_bytes); */
        /*  /1* cerr << "Workspace size: " << workspace_bytes << endl; *1/ */


        /*  /1* hipMalloc( &U, ( dN * dC * dH * dW ) * sizeof( float ) ); *1/
         */
        /*  /1* hipMalloc( &V, ( dN * dC * dH * dW ) * sizeof( float ) ); *1/
         */
        /*  /1* hipMalloc( &K, ( dC * dF * dKH * dKW ) * sizeof( float ) ); *1/
         */

        /*  hipMalloc(&d_workspace, workspace_bytes); */

        /*  const float alpha = 1, beta = 0; */
        /*  hipdnnConvolutionForward(cudnn, */
        /*                          &alpha, */
        /*                          input_descriptor, */
        /*                          U, */
        /*                          kernel_descriptor, */
        /*                          K, */
        /*                          convolution_descriptor, */
        /*                          convolution_algorithm, */
        /*                          d_workspace, */
        /*                          workspace_bytes, */
        /*                          &beta, */
        /*                          output_descriptor, */
        /*                          V); */
        /*  hipFree(d_workspace); */

        /*  /1* hipFree( U ); *1/ */
        /*  /1* hipFree( V ); *1/ */
        /*  /1* hipFree( K ); *1/ */

        /*  hipdnnDestroyTensorDescriptor(input_descriptor); */
        /*  hipdnnDestroyTensorDescriptor(output_descriptor); */
        /*  hipdnnDestroyFilterDescriptor(kernel_descriptor); */
        /*  hipdnnDestroyConvolutionDescriptor(convolution_descriptor); */
        /*  hipdnnDestroy(cudnn); */
        /*} */
