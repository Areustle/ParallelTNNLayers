#include "../NVConv2d.cuh"

int main() {

  float* In;
  float* Out;
  float* Filter;

  unsigned N     = 1;
  unsigned C     = 16;
  unsigned H     = 32;
  unsigned W     = 32;
  unsigned fK    = 16;
  unsigned fH    = 3;
  unsigned fW    = 3;

  hipMalloc(&In, (N * C * H * W));
  hipMalloc(&Filter, fK * C * fH * fW);
  hipMalloc(&Out, (N * fK * H * W));

  float milliseconds = NV::conv2d_forward_gpu(In,
                                              N,
                                              C,
                                              H,
                                              W,
                                              Filter,
                                              fK,
                                              fH,
                                              fW,
                                              Out);

  printf("Elapsed Time: %f us \n", milliseconds * 1e3);

  hipFree(In);
  hipFree(Filter);
  hipFree(Out);
}
