#include <algorithm>
#include <fstream>
#include <iostream>
#include <set>
#include <sstream>
#include <string>
#include <type_traits>

#include "../cp4Conv2d.cuh"

using namespace std;

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

template<unsigned N,
         unsigned C,
         unsigned H,
         unsigned W,
         unsigned pad,
         unsigned fK,
         unsigned fH,
         unsigned fW,
         unsigned fRank>
void profile(streambuf* output_buffer) {
  float us = CP::run_convolution<N, C, H, W, pad, fK, fH, fW, fRank>(47);

  ostream results(output_buffer);
  results << N << "," << C << "," << H << "," << W << "," << pad << "," << fK
          << "," << fH << "," << fW << "," << fRank << ", " << us << endl;
}

template<unsigned rank> void profile_helper(streambuf* output_buffer) {

  profile_helper<rank - 1>(output_buffer);

  profile<1, 3, 512, 512, 1, 1, 3, 3, rank>(output_buffer);
  profile<2, 3, 512, 512, 1, 1, 3, 3, rank>(output_buffer);
  profile<4, 3, 512, 512, 1, 1, 3, 3, rank>(output_buffer);
  profile<8, 3, 512, 512, 1, 1, 3, 3, rank>(output_buffer);
  profile<16, 3, 512, 512, 1, 1, 3, 3, rank>(output_buffer);
  profile<32, 3, 512, 512, 1, 1, 3, 3, rank>(output_buffer);
  profile<64, 3, 512, 512, 1, 1, 3, 3, rank>(output_buffer);
  profile<128, 3, 512, 512, 1, 1, 3, 3, rank>(output_buffer);
}

template<> void profile_helper<0>(streambuf* output_buffer) {}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

int main(int argc, char** argv) {

  streambuf* output_buffer = std::cout.rdbuf();
  ofstream   of;
  int        device = 0;

  switch (argc) {
    case 3: device = atoi(argv[2]);
    case 2: of.open(argv[1]); output_buffer = of.rdbuf();
    case 1: break;
    default:
      cerr << "USAGE: BenchCP4 "
              " [Results_file] "
              " [device_number]"
           << endl;
      return 1;
  }

  hipSetDevice(device);

  ostream results(output_buffer);
  results << "N,C,H,W,pad,fK,fH,fW,fRank,us" << endl;

  profile_helper<16>(output_buffer);
}
