#include <algorithm>
#include <fstream>
#include <iostream>
#include <set>
#include <sstream>
#include <string>

#include "../cp4Conv2d.cuh"

using namespace std;


////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
template<unsigned N,
         unsigned C,
         unsigned H,
         unsigned W,
         unsigned pad,
         unsigned fK,
         unsigned fH,
         unsigned fW,
         unsigned fRank>
void profile(streambuf* output_buffer) {

  float us = CP::run_convolution<N, C, H, W, pad, fK, fH, fW, fRank>(47);

  ostream results(output_buffer);
  results << N << "," << C << "," << H << "," << W << "," << pad << "," << fK
          << "," << fH << "," << fW << "," << fRank << ", " << us << endl;
}

template<unsigned rank> void profile_helper(streambuf* output_buffer) {

  profile_helper<rank - 1>(output_buffer);

  profile<1, 3, 32,   32,   1, 1, 3, 3, rank>(output_buffer);
  profile<1, 3, 64,   64,   1, 1, 3, 3, rank>(output_buffer);
  profile<1, 3, 128,  128,  1, 1, 3, 3, rank>(output_buffer);
  profile<1, 3, 256,  256,  1, 1, 3, 3, rank>(output_buffer);
  profile<1, 3, 512,  512,  1, 1, 3, 3, rank>(output_buffer);
  profile<1, 3, 1024, 1024, 1, 1, 3, 3, rank>(output_buffer);
  profile<1, 3, 2048, 2048, 1, 1, 3, 3, rank>(output_buffer);
  profile<1, 3, 4096, 4096, 1, 1, 3, 3, rank>(output_buffer);
}

template<> void profile_helper<0>(streambuf* output_buffer) {}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

int main(int argc, char** argv) {

  streambuf* output_buffer = std::cout.rdbuf();
  ofstream   of;
  int        device = 0;

  switch (argc) {
    case 3: device = atoi(argv[2]);
    case 2: of.open(argv[1]); output_buffer = of.rdbuf();
    case 1: break;
    default:
      cerr << "USAGE: BenchCP4 "
              " [Results_file] "
              " [device_number]"
           << endl;
      return 1;
  }

  hipSetDevice(device);

  ostream results(output_buffer);
  results << "N,C,H,W,pad,fK,fH,fW,fRank,us" << endl;

  profile_helper<16>(output_buffer);
}
