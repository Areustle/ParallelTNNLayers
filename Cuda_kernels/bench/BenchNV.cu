#include <algorithm>
#include <fstream>
#include <iostream>
#include <set>
#include <sstream>
#include <string>

#include "../NVConv2d.cuh"

using namespace std;

vector<tensor_shape> get_unique_ordered_shapes(vector<tensor_shape> input) {

  vector<tensor_shape> output;
  set<tensor_shape>    seen;

  for (auto& shape : input) {
    if (seen.count(shape) == 0) output.push_back(shape);
    seen.insert(shape);
  }

  return output;
}

int main(int argc, char** argv) {

  ifstream   tensors(argv[1]);
  streambuf* output_buffer = std::cout.rdbuf();
  ofstream   of;
  int        device = 0;

  switch (argc) {
    case 4: device = atoi(argv[3]);
    case 3: of.open(argv[2]); output_buffer = of.rdbuf();
    case 2: break;
    default:
      cerr << "USAGE: BenchNV "
              " Tensor_file "
              " [Results_file] "
              " [device_number]"
           << endl;
      return 1;
  }

  ostream results(output_buffer);
  results << "N,C,H,W,pad,fK,fH,fW,us" << endl;


  if (!tensors.is_open()) {
    cerr << "Couldn't open tensors file.\n";
    return 1;
  }

  vector<tensor_shape> shapes;

  string line;

  while (getline(tensors, line)) {

    if (line[0] == '#' || line.empty()) continue;

    stringstream line_sm(line);
    unsigned     N, H, W, C, pad, fK, fH, fW, fRank;
    line_sm >> N >> C >> H >> W >> pad >> fK >> fH >> fW >> fRank;

    tensor_shape params;
    params.N     = N;
    params.C     = C;
    params.H     = H;
    params.W     = W;
    params.pad   = pad;
    params.fRank = 0;
    params.fK    = fK;
    params.fH    = fH;
    params.fW    = fW;

    shapes.push_back(params);
  }

  shapes = get_unique_ordered_shapes(shapes);

  hipSetDevice(device);

  for (auto& p : shapes) {
    float us = NV::run_convolution(p, 47);
    results << p.N << "," << p.C << "," << p.H << "," << p.W << "," << p.pad
            << "," << p.fK << "," << p.fH << "," << p.fW << "," << us << endl;
  }
}
