
#include <hip/hip_runtime.h>

__constant__ float const_filter[4096];

// The Full convolution kernel.
template<unsigned TileFactor = 1>
__global__ void conv2d_full_kernel(const float* __restrict__ Input,
                                   const unsigned pad,
                                   const unsigned fK,
                                   const unsigned fH,
                                   const unsigned fW,
                                   const unsigned C,
                                   float* __restrict__ Out) {

  extern __shared__ float shared_mem[];

  // Declare useful constants. This should be cleaned up if
  // Register pressure grows too high.
  const unsigned n         = blockIdx.z / fK;
  const unsigned k         = blockIdx.z % fK;
  const unsigned w         = threadIdx.x;
  const unsigned h         = threadIdx.y;
  const unsigned Bw        = blockDim.x;
  const unsigned Bh        = blockDim.y;
  const unsigned oW        = gridDim.x * blockDim.x * TileFactor;
  const unsigned oH        = gridDim.y * blockDim.y;
  const unsigned iW        = gridDim.x * blockDim.x * TileFactor + pad;
  const unsigned iH        = gridDim.y * blockDim.y + pad;
  const unsigned hBlockOff = blockIdx.y * blockDim.y;
  const unsigned wBlockOff = blockIdx.x * blockDim.x * TileFactor;
  const unsigned jEnd      = fH - 1 + Bh;
  const unsigned iEnd      = fW - 1 + Bw;
  const unsigned sH        = fH - 1 + Bh;
  const unsigned sW        = fW - 1 + Bw * TileFactor;

  // Shift the Global pounsigneders to our Region Of unsignederest
  Input += n * C * iH * iW  // batch number offset for this thread
           + hBlockOff * iW // h offset for this thread
           + wBlockOff;     // w offset for this thread

  Out += n * fK * oH * oW // batch offset
         + k * oH * oW    // conv filter offset
         + hBlockOff * oW // h offset
         + wBlockOff;     // w offset
  // clang-format off

  // Cooperatively load all input segment unsignedo our shared memory.
  for (unsigned c = 0; c < C; ++c)         // For every channel
  for (unsigned j = h; j < jEnd; j += Bh)  // For every participating h pixel
  for (unsigned i = w; i < iEnd; i += Bw)  // For every participating w pixel
  #pragma unroll
  for (unsigned t = 0; t < TileFactor; ++t)
    shared_mem[c*sH*sW + j*sW + i+(t*Bw)] = Input[c*iH*iW + j*iW + i+(t*Bw)];

  __syncthreads();

  // Build sum by tiling factor
  float sum[TileFactor];
  #pragma unroll
  for (unsigned t = 0; t < TileFactor; ++t) sum[t] = 0.0f;

  // Perform Convolution from shared memory
  // currently expect this to have bank conflicts. Requires padding.
  for (unsigned c = 0; c < C; ++c)
  for (unsigned r = 0; r < fH; ++r)
  for (unsigned s = 0; s < fW; ++s)
  #pragma unroll
  for (unsigned t = 0; t < TileFactor; ++t)
    sum[t] += shared_mem[c*sH*sW + (h+r)*sW + (w+s+(t*Bw))]
      * const_filter[k*C*fH*fW + c*fH*fW + r*fW + s];

  // populate output array.
  #pragma unroll
  for (unsigned t = 0; t < TileFactor; ++t)
    Out[h*oW + w+(t*Bw)] = sum[t];

  // clang-format on
}

int main() {

  float* U;
  float* V;
  void*  K;

  unsigned N  = 1;
  unsigned C  = 16;
  unsigned H  = 32;
  unsigned W  = 32;
  unsigned fK = 16;
  unsigned fH = 3;
  unsigned fW = 3;

  hipMalloc(&U, (N * C * (H + 2) * (W + 2)) * sizeof(float));
  hipMalloc(&K, (fK * C * fH * fW) * sizeof(float));
  hipMalloc(&V, (N * fK * H * W) * sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(const_filter), K, sizeof(float) * (C * fK * fH * fW));

  static const unsigned tf   = 2;
  const unsigned        bdim = 16;
  const size_t          smsz = C             //
                      * (fW - 1 + bdim * tf) //
                      * (fH - 1 + bdim) *    //
                      sizeof(float);

  const dim3 Gshp(W / (bdim * tf), H / (bdim), fK * N);
  const dim3 Bshp(bdim, bdim, 1);

  /* for (unsigned i = 0; i < 1000; ++i) { */
  conv2d_full_kernel<tf><<<Gshp, Bshp, smsz>>>(U, 2, fK, fH, fW, C, V);
  hipDeviceSynchronize();
  /* } */

  hipFree(U);
  hipFree(V);
  hipFree(K);
}
