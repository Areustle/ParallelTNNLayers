#include "../conv.cuh"

int main() {

  float* U;
  float* V;
  float* K;

  unsigned N   = 1;
  unsigned C   = 16;
  unsigned H   = 32;
  unsigned W   = 32;
  unsigned fK  = 16;
  unsigned fH  = 3;
  unsigned fW  = 3;
  unsigned pad = 1;

  hipMalloc(&U, (N * C * (H + 2 * pad) * (W + 2 * pad)) * sizeof(float));
  hipMalloc(&K, (fK * C * fH * fW) * sizeof(float));
  hipMalloc(&V, (N * fK * H * W) * sizeof(float));

  cuda_conv2d_full_gpu(U, N, C, H, W, pad, K, fK, C, fH, fW, V);

  hipFree(U);
  hipFree(V);
  hipFree(K);
}
