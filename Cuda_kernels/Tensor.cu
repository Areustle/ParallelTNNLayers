#include "Tensor.cuh"

#include <cstring>
#include <numeric>

using namespace std;

Tensor::Tensor(std::initializer_list<int> l) : shape(l) {
  const int len =
      std::accumulate(shape.begin(), shape.end(), 1, std::multiplies<int>());
  hipMallocManaged(&m_data, len * sizeof(float));
  /* memset(m_data, 0, len); */
  for (int i = 0; i < len; ++i)
    m_data[i] = 0;
  hipDeviceSynchronize();
}

Tensor::Tensor(Tensor const &other) : shape(other.shape) {
  hipMallocManaged(&m_data, other.size() * sizeof(float));
  for (int i = 0; i < other.size(); ++i)
    m_data[i] = other.m_data[i];
  hipDeviceSynchronize();
}

Tensor &Tensor::operator=(const Tensor &other) {
  if (this == &other)
    return *this;
  if (this->size() != other.size()) {
    delete[] m_data;
    hipMallocManaged(&m_data, other.size() * sizeof(float));
  }
  for (int i = 0; i < other.size(); ++i)
    m_data[i] = other.m_data[i];
  hipDeviceSynchronize();
  return *this;
}

Tensor::~Tensor() {
  hipDeviceSynchronize();
  hipFree(m_data);
}

size_t Tensor::size() const {
  return std::accumulate(shape.begin(), shape.end(), 1, std::multiplies<int>());
}
