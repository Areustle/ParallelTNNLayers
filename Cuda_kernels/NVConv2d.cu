#include "NVConv2d.cuh"
#include <hipDNN.h>
#include <iostream>
#include <stdlib.h>

using namespace std;

#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

float conv2d_forward_gpu(tensor_shape params,
                         float*       In,
                         float*       Filter,
                         float*       Out,
                         unsigned     PROFCOUNT = 1) {

  const unsigned N   = params.N;
  const unsigned C   = params.C;
  const unsigned H   = params.H;
  const unsigned W   = params.W;
  const unsigned pad = params.pad;
  const unsigned fK  = params.fK;
  const unsigned fH  = params.fH;
  const unsigned fW  = params.fW;

  hipdnnHandle_t cudnn;
  hipdnnCreate(&cudnn);

  hipdnnTensorDescriptor_t input_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(
      input_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, H, W));

  hipdnnFilterDescriptor_t kernel_descriptor;
  checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
  checkCUDNN(hipdnnSetFilter4dDescriptor(
      kernel_descriptor, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, fK, C, fH, fW));

  hipdnnConvolutionDescriptor_t convolution_descriptor;
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                             /*pad_height=*/pad,
                                             /*pad_width=*/pad,
                                             /*vertical_stride=*/1,
                                             /*horizontal_stride=*/1,
                                             /*dilation_height=*/1,
                                             /*dilation_width=*/1,
                                             /*mode=*/HIPDNN_CROSS_CORRELATION,
                                             /*computeType=*/HIPDNN_DATA_FLOAT));

  int batch_size{ 0 }, channels{ 0 }, height{ 0 }, width{ 0 };
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor,
                                                   input_descriptor,
                                                   kernel_descriptor,
                                                   &batch_size,
                                                   &channels,
                                                   &height,
                                                   &width));

  hipdnnTensorDescriptor_t output_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        batch_size,
                                        channels,
                                        height,
                                        width));


  hipdnnConvolutionFwdAlgo_t convolution_algorithm;
  checkCUDNN(
      hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                          input_descriptor,
                                          kernel_descriptor,
                                          convolution_descriptor,
                                          output_descriptor,
                                          HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                          /*memoryLimitInBytes=*/0,
                                          &convolution_algorithm));

  size_t workspace_bytes{ 0 };
  checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                     input_descriptor,
                                                     kernel_descriptor,
                                                     convolution_descriptor,
                                                     output_descriptor,
                                                     convolution_algorithm,
                                                     &workspace_bytes));

  void* d_workspace{ nullptr };
  hipMalloc(&d_workspace, workspace_bytes);

  const float alpha = 1, beta = 0;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float cumulativeNS = 0.0f;
  for (unsigned i = 0; i < PROFCOUNT; ++i) {
    hipDeviceSynchronize();
    hipEventRecord(start);
    hipdnnConvolutionForward(cudnn,
                            &alpha,
                            input_descriptor,
                            In,
                            kernel_descriptor,
                            Filter,
                            convolution_descriptor,
                            convolution_algorithm,
                            d_workspace,
                            workspace_bytes,
                            &beta,
                            output_descriptor,
                            Out);
    hipDeviceSynchronize();
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cumulativeNS += milliseconds * 1e6;
  }
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(d_workspace);
  hipdnnDestroyTensorDescriptor(input_descriptor);
  hipdnnDestroyTensorDescriptor(output_descriptor);
  hipdnnDestroyFilterDescriptor(kernel_descriptor);
  hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
  hipdnnDestroy(cudnn);

  return (cumulativeNS / PROFCOUNT);
}


/*******************************************************************************
 * Unified memory Tensorized call of Convolution
 ******************************************************************************/
Tensor NV::Conv2dForward(const Tensor In, const Tensor K, unsigned pad) {

  tensor_shape params;
  params.N   = In.shape[0];
  params.C   = In.shape[1];
  params.H   = In.shape[2];
  params.W   = In.shape[3];
  params.pad = pad;
  params.fK  = K.shape[0];
  params.fH  = K.shape[2];
  params.fW  = K.shape[3];

  Tensor V({ In.shape[0], K.shape[0], In.shape[2], In.shape[3] });
  conv2d_forward_gpu(params, In.m_data, K.m_data, V.m_data, 1);

  return V;
}


/*******************************************************************************
 * run_convolution operation with a profile count loop
 ******************************************************************************/
float NV::run_convolution(tensor_shape p, unsigned PROFCOUNT) {

  float* In;
  float* Out;
  float* Filter;


  hipMalloc(&In, p.N * p.C * p.H * p.W * sizeof(float));
  hipMalloc(&Filter, p.fK * p.C * p.fH * p.fW * sizeof(float));
  hipMalloc(&Out, p.N * p.fK * p.H * p.W * sizeof(float));

  float ns = conv2d_forward_gpu(p, In, Filter, Out, PROFCOUNT);

  hipFree(In);
  hipFree(Filter);
  hipFree(Out);

  return ns;
}


/*******************************************************************************
 * Main function. call 1 instance of kernel execution
 ******************************************************************************/
int main(int argc, char** argv) {

  unsigned N   = 5;
  unsigned C   = 32;
  unsigned H   = 1024;
  unsigned W   = 1024;
  unsigned pad = 1;
  unsigned fK  = 32;
  unsigned fH  = 3;
  unsigned fW  = 3;

  if (argc != 11) {
    hipSetDevice(0);
    cerr << "Using default shape" << endl;
  } else {
    N   = atoi(argv[1]);
    C   = atoi(argv[2]);
    H   = atoi(argv[3]);
    W   = atoi(argv[4]);
    pad = atoi(argv[5]);
    fK  = atoi(argv[6]);
    fH  = atoi(argv[7]);
    fW  = atoi(argv[8]);
    // fRank var meaningless here
    hipSetDevice(atoi(argv[10]));
  }

  tensor_shape params;
  params.N   = N;
  params.C   = C;
  params.H   = H;
  params.W   = W;
  params.pad = pad;
  params.fK  = fK;
  params.fH  = fH;
  params.fW  = fW;

  NV::run_convolution(params, 1);
}
