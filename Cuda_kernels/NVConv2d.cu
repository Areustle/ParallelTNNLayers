#include "NVConv2d.cuh"
#include <hipDNN.h>
#include <iostream>
#include <stdlib.h>

using namespace std;

#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

void NV::conv2d_forward_gpu(float*   In,
                            unsigned N,
                            unsigned C,
                            unsigned H,
                            unsigned W,
                            unsigned pad,
                            float*   Filter,
                            unsigned fK,
                            unsigned fH,
                            unsigned fW,
                            float*   Out) {
  hipdnnHandle_t cudnn;
  hipdnnCreate(&cudnn);

  hipdnnTensorDescriptor_t input_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(
      input_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, H, W));

  hipdnnFilterDescriptor_t kernel_descriptor;
  checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
  checkCUDNN(hipdnnSetFilter4dDescriptor(
      kernel_descriptor, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, fK, C, fH, fW));

  hipdnnConvolutionDescriptor_t convolution_descriptor;
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                             /*pad_height=*/pad,
                                             /*pad_width=*/pad,
                                             /*vertical_stride=*/1,
                                             /*horizontal_stride=*/1,
                                             /*dilation_height=*/1,
                                             /*dilation_width=*/1,
                                             /*mode=*/HIPDNN_CROSS_CORRELATION,
                                             /*computeType=*/HIPDNN_DATA_FLOAT));

  int batch_size{ 0 }, channels{ 0 }, height{ 0 }, width{ 0 };
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor,
                                                   input_descriptor,
                                                   kernel_descriptor,
                                                   &batch_size,
                                                   &channels,
                                                   &height,
                                                   &width));

  hipdnnTensorDescriptor_t output_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        batch_size,
                                        channels,
                                        height,
                                        width));


  hipdnnConvolutionFwdAlgo_t convolution_algorithm;
  checkCUDNN(
      hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                          input_descriptor,
                                          kernel_descriptor,
                                          convolution_descriptor,
                                          output_descriptor,
                                          HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                          /*memoryLimitInBytes=*/0,
                                          &convolution_algorithm));

  size_t workspace_bytes{ 0 };
  checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                     input_descriptor,
                                                     kernel_descriptor,
                                                     convolution_descriptor,
                                                     output_descriptor,
                                                     convolution_algorithm,
                                                     &workspace_bytes));

  void* d_workspace{ nullptr };
  hipMallocManaged(&d_workspace, workspace_bytes);

  float* d_input{ nullptr };
  hipMalloc(&d_input, N * C * H * W);
  hipMemcpy(d_input, In, N * C * H * W, hipMemcpyHostToDevice);

  size_t out_bytes = batch_size * channels * height * width * sizeof(float);
  float* d_output{ nullptr };
  hipMalloc(&d_output, out_bytes);
  hipMemset(d_output, 0, out_bytes);

  const float alpha = 1, beta = 0;
  hipdnnConvolutionForward(cudnn,
                          &alpha,
                          input_descriptor,
                          In,
                          kernel_descriptor,
                          Filter,
                          convolution_descriptor,
                          convolution_algorithm,
                          d_workspace,
                          workspace_bytes,
                          &beta,
                          output_descriptor,
                          Out);

  hipFree(d_workspace);
  hipdnnDestroyTensorDescriptor(input_descriptor);
  hipdnnDestroyTensorDescriptor(output_descriptor);
  hipdnnDestroyFilterDescriptor(kernel_descriptor);
  hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
  hipdnnDestroy(cudnn);
}

Tensor NV::Conv2dForward(const Tensor In, const Tensor K, unsigned pad) {

  Tensor V({ In.shape[0], K.shape[0], In.shape[2], In.shape[3] });
  NV::conv2d_forward_gpu(In.m_data,
                         In.shape[0],
                         In.shape[1],
                         In.shape[2],
                         In.shape[3],
                         pad,
                         K.m_data,
                         K.shape[0],
                         K.shape[2],
                         K.shape[3],
                         V.m_data);

  return V;
}


int main(int argc, char** argv) {

  unsigned N   = 5;
  unsigned C   = 32;
  unsigned H   = 1024;
  unsigned W   = 1024;
  unsigned pad = 1;
  unsigned fK  = 32;
  unsigned fH  = 3;
  unsigned fW  = 3;

  /* int devcnt = -1; */

  if (argc != 11) {
    hipSetDevice(0);
    cerr << "Using default shape" << endl;
  } else {
    N   = atoi(argv[1]);
    C   = atoi(argv[2]);
    H   = atoi(argv[3]);
    W   = atoi(argv[4]);
    pad = atoi(argv[5]);
    fK  = atoi(argv[6]);
    fH  = atoi(argv[7]);
    fW  = atoi(argv[8]);
    // fRank var meaningless here
    hipSetDevice(atoi(argv[10]));
    /* cerr << atoi(argv[10]) << " " << argv[10] << endl; */
  }

  /* cerr << hipGetDevice(&devcnt) << endl; */
  /* cerr << devcnt << endl; */

  float* In;
  float* Out;
  float* Filter;


  hipMalloc(&In, N * C * H * W * sizeof(float));
  hipMalloc(&Filter, fK * C * fH * fW * sizeof(float));
  hipMalloc(&Out, N * fK * H * W * sizeof(float));

  NV::conv2d_forward_gpu(In, N, C, H, W, pad, Filter, fK, fH, fW, Out);

  hipFree(In);
  hipFree(Filter);
  hipFree(Out);
}
