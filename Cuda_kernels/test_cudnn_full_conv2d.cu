#include <hipDNN.h>
#include <iostream>

using namespace std;

int main() {

  size_t PROFCOUNT = 100000;

  float* U;
  void*  K;
  float* V;
  void*  d_workspace;

  const size_t dN = 1, dC = 16, dH = 32, dW = 32, dKN = 16, dKH = 3, dKW = 3;

  hipdnnHandle_t                cudnn;
  hipdnnTensorDescriptor_t      input_descriptor;
  hipdnnTensorDescriptor_t      output_descriptor;
  hipdnnFilterDescriptor_t      kernel_descriptor;
  hipdnnConvolutionDescriptor_t convolution_descriptor;
  hipdnnConvolutionFwdAlgo_t    convolution_algorithm;
  size_t                       workspace_bytes = 0;

  hipdnnCreate( &cudnn );
  hipdnnCreateTensorDescriptor( &input_descriptor );
  hipdnnSetTensor4dDescriptor(
      input_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, dN, dC, dH, dW );
  hipdnnCreateTensorDescriptor( &output_descriptor );
  hipdnnSetTensor4dDescriptor(
      output_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, dN, dC, dH, dW );
  hipdnnCreateFilterDescriptor( &kernel_descriptor );
  hipdnnSetFilter4dDescriptor( kernel_descriptor,
                              HIPDNN_DATA_FLOAT,
                              HIPDNN_TENSOR_NCHW,
                              dC,
                              dKN,
                              dKH,
                              dKW );
  hipdnnCreateConvolutionDescriptor( &convolution_descriptor );
  hipdnnSetConvolution2dDescriptor( convolution_descriptor,
                                   1,
                                   1,
                                   1,
                                   1,
                                   1,
                                   1,
                                   HIPDNN_CROSS_CORRELATION,
                                   HIPDNN_DATA_FLOAT );
  hipdnnGetConvolutionForwardAlgorithm( cudnn,
                                       input_descriptor,
                                       kernel_descriptor,
                                       convolution_descriptor,
                                       output_descriptor,
                                       HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                       /*memoryLimitInBytes=*/0,
                                       &convolution_algorithm );
  hipdnnGetConvolutionForwardWorkspaceSize( cudnn,
                                           input_descriptor,
                                           kernel_descriptor,
                                           convolution_descriptor,
                                           output_descriptor,
                                           convolution_algorithm,
                                           &workspace_bytes );
  cerr << "Workspace size: " << workspace_bytes << endl;


  hipMalloc( &U, ( 1 * 16 * 32 * 32 ) * sizeof( float ) );
  hipMalloc( &V, ( 1 * 16 * 32 * 32 ) * sizeof( float ) );
  hipMalloc( &d_workspace, workspace_bytes );
  hipMalloc( &K, ( 16 * 16 * 3 * 3 ) * sizeof( float ) );


  const float alpha = 1, beta = 0;
  for ( int i = 0; i < PROFCOUNT; ++i )
    hipdnnConvolutionForward( cudnn,
                             &alpha,
                             input_descriptor,
                             U,
                             kernel_descriptor,
                             K,
                             convolution_descriptor,
                             convolution_algorithm,
                             d_workspace,
                             workspace_bytes,
                             &beta,
                             output_descriptor,
                             V );

  hipFree( U );
  hipFree( V );
  hipFree( K );
  hipFree( d_workspace );

  hipdnnDestroyTensorDescriptor( input_descriptor );
  hipdnnDestroyTensorDescriptor( output_descriptor );
  hipdnnDestroyFilterDescriptor( kernel_descriptor );
  hipdnnDestroyConvolutionDescriptor( convolution_descriptor );
  hipdnnDestroy( cudnn );
}
