#include "CudaAllocator.cuh"

float* CudaAllocator::allocate(std::size_t n) {
  float* p;
  hipMalloc(&p, n * sizeof(float));
  hipDeviceSynchronize();
  return p;
}

void CudaAllocator::deallocate(float* p, std::size_t n) {
  hipDeviceSynchronize();
  hipFree(p);
}


