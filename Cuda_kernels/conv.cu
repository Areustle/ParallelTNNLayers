#include "hip/hip_runtime.h"
#include "conv.cuh"

__constant__ float const_filter[4096];

// The Full convolution kernel.
template<unsigned TileFactor = 1>
__global__ void conv2d_full_kernel(const float* __restrict__ Input,
                                   const unsigned pad,
                                   const unsigned fK,
                                   const unsigned fH,
                                   const unsigned fW,
                                   const unsigned C,
                                   float* __restrict__ Out) {

  extern __shared__ float shared_mem[];

  // Declare useful constants. This should be cleaned up if
  // Register pressure grows too high.
  const unsigned n         = blockIdx.z / fK;
  const unsigned k         = blockIdx.z % fK;
  const unsigned w         = threadIdx.x;
  const unsigned h         = threadIdx.y;
  const unsigned Bw        = blockDim.x;
  const unsigned Bh        = blockDim.y;
  const unsigned oW        = gridDim.x * blockDim.x * TileFactor;
  const unsigned oH        = gridDim.y * blockDim.y;
  const unsigned iW        = gridDim.x * blockDim.x * TileFactor + pad;
  const unsigned iH        = gridDim.y * blockDim.y + pad;
  const unsigned hBlockOff = blockIdx.y * blockDim.y;
  const unsigned wBlockOff = blockIdx.x * blockDim.x * TileFactor;
  const unsigned jEnd      = fH - 1 + Bh;
  const unsigned iEnd      = fW - 1 + Bw;
  const unsigned sH        = fH - 1 + Bh;
  const unsigned sW        = fW - 1 + Bw * TileFactor;

  // Shift the Global pounsigneders to our Region Of unsignederest
  Input += n * C * iH * iW  // batch number offset for this thread
           + hBlockOff * iW // h offset for this thread
           + wBlockOff;     // w offset for this thread

  Out += n * fK * oH * oW // batch offset
         + k * oH * oW    // conv filter offset
         + hBlockOff * oW // h offset
         + wBlockOff;     // w offset
  // clang-format off

  // Cooperatively load all input segment unsignedo our shared memory.
  for (unsigned c = 0; c < C; ++c)         // For every channel
  for (unsigned j = h; j < jEnd; j += Bh)  // For every participating h pixel
  for (unsigned i = w; i < iEnd; i += Bw)  // For every participating w pixel
  #pragma unroll
  for (unsigned t = 0; t < TileFactor; ++t)
    shared_mem[c*sH*sW + j*sW + i+(t*Bw)] = Input[c*iH*iW + j*iW + i+(t*Bw)];

  __syncthreads();

  // Build sum by tiling factor
  float sum[TileFactor];
  #pragma unroll
  for (unsigned t = 0; t < TileFactor; ++t) sum[t] = 0.0f;

  // Perform Convolution from shared memory
  // currently expect this to have bank conflicts. Requires padding.
  for (unsigned c = 0; c < C; ++c)
  for (unsigned r = 0; r < fH; ++r)
  for (unsigned s = 0; s < fW; ++s)
  #pragma unroll
  for (unsigned t = 0; t < TileFactor; ++t)
    sum[t] += shared_mem[c*sH*sW + (h+r)*sW + (w+s+(t*Bw))]
      * const_filter[k*C*fH*fW + c*fH*fW + r*fW + s];

  // populate output array.
  #pragma unroll
  for (unsigned t = 0; t < TileFactor; ++t)
    Out[h*oW + w+(t*Bw)] = sum[t];

  // clang-format on
}


Tensor conv2d_full_gpu(Tensor const Input, Tensor const Filter, int pad) {

  const int N  = Input.shape[0];
  const int C  = Input.shape[1];
  const int H  = Input.shape[2] - 2 * pad;
  const int W  = Input.shape[3] - 2 * pad;
  const int fK = Filter.shape[0];
  const int FC = Filter.shape[1];
  const int fH = Filter.shape[2];
  const int fW = Filter.shape[3];

  Tensor Out{ N, fK, H, W };

  hipMemcpyToSymbol(HIP_SYMBOL(
      const_filter), Filter.m_data, sizeof(float) * Filter.size());

  static const int tf   = 2;
  const int        bdim = 16;
  const size_t     smsz = C                  //
                      * (fW - 1 + bdim * tf) //
                      * (fH - 1 + bdim) *    //
                      sizeof(float);

  const dim3 Gshp(W / (bdim * tf), H / (bdim), fK * N);
  const dim3 Bshp(bdim, bdim, 1);

  conv2d_full_kernel<tf>
      <<<Gshp, Bshp, smsz>>>(Input.m_data, 2 * pad, fK, fH, fW, C, Out.m_data);
  hipDeviceSynchronize();

  return Out;
}

Tensor conv2d_full_cpu(Tensor const Input, Tensor const Filter) {

  const int N  = Input.shape[0];
  const int C  = Input.shape[1];
  const int H  = Input.shape[2];
  const int W  = Input.shape[3];
  const int FK = Filter.shape[0];
  const int FC = Filter.shape[1];
  const int FR = Filter.shape[2];
  const int FS = Filter.shape[3];

  const int FRCenter = FR / 2;
  const int FSCenter = FS / 2;

  Tensor Out{ N, C, H, W };

  // clang-format off
  for (int n = 0; n < N; ++n)
  for (int fk = 0; fk < FK; ++fk)
  for (int h = 0; h < H; ++h)
  for (int w = 0; w < W; ++w){
    float sum = 0.0f;
    for (int c = 0; c < C; ++c)
    for (int fr = 0; fr < FR; ++fr)
    for (int fs = 0; fs < FS; ++fs){

      const int hIdx = h + (fr - FRCenter);
      const int wIdx = w + (fs - FSCenter);

      if(hIdx >= 0 && hIdx < H && wIdx >= 0 && wIdx < W){
            sum += Input.m_data[n*C*H*W + c*H*W + hIdx*W + wIdx]
            *  Filter.m_data[fk*C*FR*FS + c*FR*FS + fr*FS + fs];
      }

    }
    Out.m_data[n*C*H*W + fk*H*W + h*W + w] = sum;
  }
  // clang-format on

  return Out;
}
