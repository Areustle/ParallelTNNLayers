#include "hip/hip_runtime.h"
#include "conv.cuh"

__constant__ float const_filter[4096];

template<int TileFactor = 1>
__global__ void conv2d_full_kernel(const float *__restrict__ Input,
                                   const int pad,
                                   const int fH,
                                   const int fW,
                                   const int C,
                                   float *__restrict__ Out) {

  extern __shared__ float shared_mem[];

  // Declare useful constants. This should be cleaned up if
  // Register pressure grows too high.
  const int w         = threadIdx.x;
  const int h         = threadIdx.y;
  const int k         = threadIdx.z * blockDim.z + blockIdx.z;
  const int Bw        = blockDim.x;
  const int Bh        = blockDim.y;
  const int oW        = gridDim.x * blockDim.x * TileFactor;
  const int oH        = gridDim.y * blockDim.y;
  const int iW        = gridDim.x * blockDim.x * TileFactor + pad;
  const int iH        = gridDim.y * blockDim.y + pad;
  const int hBlockOff = blockIdx.y * blockDim.y;
  const int wBlockOff = blockIdx.x * blockDim.x * TileFactor;

  // Shift the Input pointer to our Region Of Interest
  Input += hBlockOff * iW + wBlockOff;
  Out += hBlockOff * oW + wBlockOff;

  // Cooperatively load all input segment into our shared memory.
  const int jEnd = fH - 1 + Bh;
  const int iEnd = fW - 1 + Bw;
  const int sH   = fH - 1 + Bh;
  const int sW   = fW - 1 + Bw * TileFactor;
  // clang-format off
  for (int c = 0; c < C; ++c)
  for (int j = h; j < jEnd; j += Bh)
  for (int i = w; i < iEnd; i += Bw)
  #pragma unroll
  for (int t = 0; t < TileFactor; ++t)
    shared_mem[c*sH*sW + j*sW + i+(t*Bw)] = Input[c*iH*iW + j*iW + i+(t*Bw)];

  __syncthreads();

  // Build sum by tiling factor
  float sum[TileFactor];
#pragma unroll
  for (int t = 0; t < TileFactor; ++t) sum[t] = 0.0f;

  // Perform Convolution from shared memory
  for (int c = 0; c < C; ++c)
  for (int r = 0; r < fH; ++r)
  for (int s = 0; s < fW; ++s)
  #pragma unroll
  for (int t = 0; t < TileFactor; ++t)
    /* sum[t] += shared_mem[c*sH*sW + (h+r)*sW + (w+s+(t*Bw))] */
    /*   * const_filter[k*C*fH*fW + c*fH*fW + r*fW + s]; */
    sum[t] += Input[c*iH*iW + (h+r)*iW + (w+s+(t*Bw))]
              * const_filter[k*C*fH*fW + c*fH*fW + r*fW + s];

// populate output array.
#pragma unroll
  for (int t = 0; t < TileFactor; ++t)
    Out[k*oH*oW + h*oW + w + (t*Bw)] = sum[t];

  // clang-format on
}

Tensor conv2d_full_gpu(Tensor const Input, Tensor const Filter) {

  const int N  = Input.shape[0];
  const int C  = Input.shape[1];
  const int H  = Input.shape[2] - 2;
  const int W  = Input.shape[3] - 2;
  const int FK = Filter.shape[0];
  const int FC = Filter.shape[1];
  const int fH = Filter.shape[2];
  const int fW = Filter.shape[3];

  Tensor Out{ N, FK, H, W };

  hipMemcpyToSymbol(HIP_SYMBOL(
      const_filter), Filter.m_data, sizeof(float) * Filter.size());

  static const int tf   = 1;
  const int        bdim = 1;
  const size_t     shared_mem_size =
      C * (fW - 1 + bdim * tf) * (fH - 1 + bdim) * sizeof(float);
  const dim3 gridDim0(W / (tf * bdim), H / (bdim), FK);
  const dim3 blockDim0(bdim, bdim, 1);

  conv2d_full_kernel<tf><<<gridDim0, blockDim0, shared_mem_size>>>(
      Input.m_data, 2, fH, fW, C, Out.m_data);
  hipDeviceSynchronize();

  return Out;
}

Tensor conv2d_full_cpu(Tensor const Input, Tensor const Filter) {

  const int N  = Input.shape[0];
  const int C  = Input.shape[1];
  const int H  = Input.shape[2];
  const int W  = Input.shape[3];
  const int FK = Filter.shape[0];
  const int FC = Filter.shape[1];
  const int FR = Filter.shape[2];
  const int FS = Filter.shape[3];

  const int FRCenter = FR / 2;
  const int FSCenter = FS / 2;

  Tensor Out{ N, C, H, W };

  // clang-format off
  for (int n = 0; n < N; ++n)
  for (int fk = 0; fk < FK; ++fk)
  for (int h = 0; h < H; ++h)
  for (int w = 0; w < W; ++w){
    float sum = 0.0f;
    for (int c = 0; c < C; ++c)
    for (int fr = 0; fr < FR; ++fr)
    for (int fs = 0; fs < FS; ++fs){

      const int hIdx = h + (fr - FRCenter);
      const int wIdx = w + (fs - FSCenter);

      if(hIdx >= 0 && hIdx < H && wIdx >= 0 && wIdx < W){
            sum += Input.m_data[n*C*H*W + c*H*W + hIdx*W + wIdx]
            *  Filter.m_data[fk*C*FR*FS + c*FR*FS + fr*FS + fs];
      }

    }
    Out.m_data[n*C*H*W + fk*H*W + h*W + w] = sum;
  }
  // clang-format on

  return Out;
}
