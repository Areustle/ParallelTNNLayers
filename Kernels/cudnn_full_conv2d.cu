#include <hipDNN.h>
#include <iostream>

int main(){

  size_t PROFCOUNT = 100000;

  float* U;
  float* V;

  hipMalloc(&U, (1*16*32*32)*sizeof(float));
  hipMalloc(&V, (1*16*32*32)*sizeof(float));

  /* Begin cuDNN Full Convolution profile section */

  hipdnnHandle_t cudnn;
  hipdnnCreate(&cudnn);

  hipdnnTensorDescriptor_t input_descriptor;
  hipdnnCreateTensorDescriptor(&input_descriptor);
  hipdnnSetTensor4dDescriptor(input_descriptor, HIPDNN_TENSOR_NCHW,
                               HIPDNN_DATA_FLOAT, 1, 16, 32, 32);
  hipdnnTensorDescriptor_t output_descriptor;
  hipdnnCreateTensorDescriptor(&output_descriptor);
  hipdnnSetTensor4dDescriptor(output_descriptor, HIPDNN_TENSOR_NCHW,
                               HIPDNN_DATA_FLOAT, 1, 16, 32, 32);
  hipdnnFilterDescriptor_t kernel_descriptor;
  hipdnnCreateFilterDescriptor(&kernel_descriptor);
  hipdnnSetFilter4dDescriptor(kernel_descriptor, HIPDNN_DATA_FLOAT,
                               HIPDNN_TENSOR_NCHW, 16, 16, 3, 3);
  hipdnnConvolutionDescriptor_t convolution_descriptor;
  hipdnnCreateConvolutionDescriptor(&convolution_descriptor);
  hipdnnSetConvolution2dDescriptor(convolution_descriptor, 1, 1, 1, 1, 1, 1,
                               HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

  hipdnnConvolutionFwdAlgo_t convolution_algorithm;
  hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                      input_descriptor,
                                      kernel_descriptor,
                                      convolution_descriptor,
                                      output_descriptor,
                                      HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                      /*memoryLimitInBytes=*/0,
                                      &convolution_algorithm);
  size_t workspace_bytes = 0;
  hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                           input_descriptor,
                                           kernel_descriptor,
                                           convolution_descriptor,
                                           output_descriptor,
                                           convolution_algorithm,
                                           &workspace_bytes);
  std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB"
            << std::endl;

  void* d_workspace;
  hipMalloc(&d_workspace, workspace_bytes);

  void * K;
  hipMalloc(&K, (16*16*3*3)*sizeof(float));

  const float alpha = 1, beta = 0;
  for (int i = 0; i<PROFCOUNT; ++i){
    hipdnnConvolutionForward(cudnn, &alpha,
        input_descriptor, U,
        kernel_descriptor, K,
        convolution_descriptor, convolution_algorithm,
        d_workspace, workspace_bytes, &beta,
        output_descriptor, V);
    hipDeviceSynchronize();
}

  hipFree(U);
  hipFree(V);
  hipFree(K);
  hipFree(d_workspace);

  hipdnnDestroyTensorDescriptor(input_descriptor);
  hipdnnDestroyTensorDescriptor(output_descriptor);
  hipdnnDestroyFilterDescriptor(kernel_descriptor);
  hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
  hipdnnDestroy(cudnn);

}
