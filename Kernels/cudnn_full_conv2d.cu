#include <hipDNN.h>
#include <iostream>

int main(){

  size_t PROFCOUNT = 1;

  float* U;
  float* V;

  int N = 1;
  int C = 16;
  int H = 32;
  int W = 32;
  int fK = 16;
  int fH = 3;
  int fW = 3;

  hipMalloc(&U, (N*C*H*W)*sizeof(float));
  hipMalloc(&V, (N*fK*H*W)*sizeof(float));

  /* Begin cuDNN Full Convolution profile section */

  hipdnnHandle_t cudnn;
  hipdnnCreate(&cudnn);

  hipdnnTensorDescriptor_t input_descriptor;
  hipdnnCreateTensorDescriptor(&input_descriptor);
  hipdnnSetTensor4dDescriptor(input_descriptor, HIPDNN_TENSOR_NCHW,
                               HIPDNN_DATA_FLOAT, N, C, H, W);
  hipdnnTensorDescriptor_t output_descriptor;
  hipdnnCreateTensorDescriptor(&output_descriptor);
  hipdnnSetTensor4dDescriptor(output_descriptor, HIPDNN_TENSOR_NCHW,
                               HIPDNN_DATA_FLOAT, N, fK, H, W);
  hipdnnFilterDescriptor_t kernel_descriptor;
  hipdnnCreateFilterDescriptor(&kernel_descriptor);
  hipdnnSetFilter4dDescriptor(kernel_descriptor, HIPDNN_DATA_FLOAT,
                               HIPDNN_TENSOR_NCHW, fK, C, fH, fW);
  hipdnnConvolutionDescriptor_t convolution_descriptor;
  hipdnnCreateConvolutionDescriptor(&convolution_descriptor);
  hipdnnSetConvolution2dDescriptor(convolution_descriptor, 1, 1, 1, 1, 1, 1,
                               HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

  hipdnnConvolutionFwdAlgo_t convolution_algorithm;
  hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                      input_descriptor,
                                      kernel_descriptor,
                                      convolution_descriptor,
                                      output_descriptor,
                                      HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                      /*memoryLimitInBytes=*/0,
                                      &convolution_algorithm);
  size_t workspace_bytes = 0;
  hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                           input_descriptor,
                                           kernel_descriptor,
                                           convolution_descriptor,
                                           output_descriptor,
                                           convolution_algorithm,
                                           &workspace_bytes);

  void* d_workspace;
  hipMalloc(&d_workspace, workspace_bytes);

  void * K;
  hipMalloc(&K, (fK*C*fH*fW)*sizeof(float));

  const float alpha = 1, beta = 0;
  for (int i = 0; i<PROFCOUNT; ++i){
    hipdnnConvolutionForward(cudnn, &alpha,
        input_descriptor, U,
        kernel_descriptor, K,
        convolution_descriptor, convolution_algorithm,
        d_workspace, workspace_bytes, &beta,
        output_descriptor, V);
    hipDeviceSynchronize();
  }

  hipFree(U);
  hipFree(V);
  hipFree(K);
  hipFree(d_workspace);

  hipdnnDestroyTensorDescriptor(input_descriptor);
  hipdnnDestroyTensorDescriptor(output_descriptor);
  hipdnnDestroyFilterDescriptor(kernel_descriptor);
  hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
  hipdnnDestroy(cudnn);

}
